#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <cutil.h>
#include <sys/time.h>

#define BLOCK_SIZE 16 
#define MATRIX_SIZE 4096 
#define WINDOW_SIZE 2

bool InitCUDA(void)
{
        int count = 0;int i = 0;
        hipGetDeviceCount(&count);
        if(count == 0) {
                fprintf(stderr, "There is no device.\n");
                return false;
        }
        for(i = 0; i < count; i++) {
                hipDeviceProp_t prop;
                if(hipGetDeviceProperties(&prop, i) == hipSuccess) {
                        if(prop.major >= 1) {
                                break;
        }}}
        if(i == count) {
                fprintf(stderr, "There is no device supporting CUDA.\n");
                return false;
        }
        hipSetDevice(i);
        printf("CUDA initialized.\n");
        return true;
}

__global__ void MatMulKernel(float* Md, float* Nd, float* Pd)
{
    int tx = threadIdx.x; int ty = threadIdx.y;
    int bx = blockIdx.x; int by = blockIdx.y;

    float Pvalue = 0;
    
    for(int m = MATRIX_SIZE * BLOCK_SIZE * by, n = BLOCK_SIZE * bx;
            m <= MATRIX_SIZE * BLOCK_SIZE * by + MATRIX_SIZE -1;
            m += BLOCK_SIZE, n += BLOCK_SIZE * MATRIX_SIZE)
    {
        __shared__ float Mds[BLOCK_SIZE][BLOCK_SIZE];
        __shared__ float Nds[BLOCK_SIZE][BLOCK_SIZE];
        
        Mds[ty][tx] = Md[m + MATRIX_SIZE * ty + tx];
        Nds[ty][tx] = Nd[n + MATRIX_SIZE * ty + tx];
       
        // Make sure that all the threads have copied to shared memory before 
       // performing the actual multiplication
        __syncthreads();
        
#pragma unroll
        for( int k = 0; k < BLOCK_SIZE; ++k)
        {
            Pvalue += Mds[ty][k] * Nds[k][tx];
        }
        
        // Syncronize after the multiplication.
        __syncthreads();    
    }
    Pd[MATRIX_SIZE * BLOCK_SIZE * by + 
        BLOCK_SIZE * bx + 
        MATRIX_SIZE * ty 
        + tx] = Pvalue;
}

int main(int argc, char* argv[])
{
    struct timeval t0,t1;
    // Initialize CUDA using the ASC helper function
    if(!InitCUDA()) {
            return 0;
    }
    // Define some sizes for malloc
    unsigned int size = MATRIX_SIZE * MATRIX_SIZE;
    unsigned int mem_size = sizeof(float) * size;
    // Declare the variables to be used 
    float* A = (float*) malloc(mem_size);
    float* B = (float*) malloc(mem_size);
    float* C = (float*) malloc(mem_size);
    float* Md;
    float* Nd;
    float* Pd;
    // Initialize the A and B matricies to the homework specifications 
    int row,col;
    for( int i=0; i<size; i++)
    {
        row = i/MATRIX_SIZE;
        col = i%MATRIX_SIZE;
        A[i] = ((row + 1.0)*(col + 1.0))/MATRIX_SIZE;
        B[i] = (col + 1.0)/(row + 1.0);
    }
    
    gettimeofday(&t0,0);
    // Allocate the matricies on the video card
    hipMalloc((void**) &Md, mem_size);
    hipMalloc((void**) &Nd, mem_size);
    hipMalloc((void**) &Pd, mem_size);
    // Copy the matricies to the video card
    hipMemcpy(Md, A, mem_size, hipMemcpyHostToDevice);
    hipMemcpy(Nd, B, mem_size, hipMemcpyHostToDevice);
    // Perform the Kernel 
    dim3 dimBlock(BLOCK_SIZE,BLOCK_SIZE);
    dim3 dimGrid(MATRIX_SIZE/dimBlock.x,MATRIX_SIZE/dimBlock.y);
    MatMulKernel<<<dimGrid, dimBlock>>>(Md,Nd,Pd);
    // Copy the results
    hipMemcpy(C, Pd, mem_size, hipMemcpyDeviceToHost);
    // Clear the memory on the video card
    hipFree(Md);hipFree(Nd);hipFree(Pd);

    gettimeofday(&t1,0);
    // Print a 16x16 "test section" to prove results are correct. 
    for( int i=0; i<16; i++){
        for( int j=0; j<16; j++){
            printf("%6.2f ",C[j*MATRIX_SIZE+i]);
        }
        printf("\n");
    }

    printf("\nTime Results\n");
    float totalInt = t1.tv_sec - t0.tv_sec + (t1.tv_usec - t0.tv_usec)*1.0E-06;
    printf("Total Execution Time:\t%e\n",totalInt);

    free(A);free(B);free(C);
    return 0;
}
